#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "util.h"

// CUDA path: -L/opt/nvidia/hpc_sdk/Linux_x86_64/21.2/cuda/11.2/lib64
// nvcc -w -m64 -gencode arch=compute_72,code=sm_72 -gencode arch=compute_70,code=sm_70 -c -w kernel.cu


__global__ void kernel(int *array1, int *array2, int *array3)

{
  // to do: verify the gpu id by using function hipGetDevice()

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  array3[index] = array1[index] + array2[index];

}

void run_kernel(int processID)
{

  printf("Par is %d\n",par);
  int i, array1[6], array2[6], array3[6], *devarray1, *devarray2, *devarray3;

  for(i = 0; i < 6; i++)

  {

    array1[i] = i;

    array2[i] = i;

  }
  // select a unique GPU id based on MPI process ID
  hipSetDevice(processID);

  hipMalloc((void**) &devarray1, sizeof(int)*6);

  hipMalloc((void**) &devarray2, sizeof(int)*6);

  hipMalloc((void**) &devarray3, sizeof(int)*6);

  hipMemcpy(devarray1, array1, sizeof(int)*6, hipMemcpyHostToDevice);

  hipMemcpy(devarray2, array2, sizeof(int)*6, hipMemcpyHostToDevice);

  kernel<<<2, 3>>>(devarray1, devarray2, devarray3);

  hipMemcpy(array3, devarray3, sizeof(int)*6, hipMemcpyDeviceToHost);

  for(i = 0; i < 6; i++)
  {

   printf("%d ", array3[i]);

  }

  printf("\n");

  hipFree(devarray1);

  hipFree(devarray2);

  hipFree(devarray3);

}
